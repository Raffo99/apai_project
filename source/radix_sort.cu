
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <malloc.h>

#define NUM_BANKS 16 
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) \
	((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

#define MAX_BLOCK_SIZE 256

int checkCudaError(hipError_t error) {
    if (error != hipSuccess) {
        printf("%s", hipGetErrorString(error));
        return true;
    }

    return false;
}

__global__ void prescan(int *g_odata, int *g_idata, int n) { 
    extern __shared__ int temp[];
    int thread_id = threadIdx.x;
    int offset = 1;

    int ai = thread_id; 
    int bi = thread_id + (n/2); 
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai); 
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    temp[ai + bankOffsetA] = g_idata[ai]; 
    temp[bi + bankOffsetB] = g_idata[bi]; 

    for (int d = n>>1; d > 0; d >>= 1) {
        __syncthreads();
        if (thread_id < d) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    } 

    if (thread_id == 0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; }

    for (int d = 1; d < n; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (thread_id < d) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            int t = temp[ai]; 
            temp[ai] = temp[bi]; 
            temp[bi] += t;
        }
    }
    __syncthreads();

    g_odata[ai] = temp[ai + bankOffsetA]; 
    g_odata[bi] = temp[bi + bankOffsetB]; 
}

__global__ void add_block_sums(int* const out, int* const in, int* const block_sums, const size_t number_elements) {
    int block_sums_value = block_sums[blockIdx.x];
    unsigned int work_id = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    if (work_id < number_elements) {
        out[work_id] = in[work_id] + block_sums_value;
        if (work_id + blockDim.x < number_elements) out[work_id + blockDim.x] = in[work_id + blockDim.x] + block_sums_value;
    }
}

void sum_scan_blelloch(int* const out, int* const in, const size_t number_elements) {
    if (checkCudaError(hipMemset(out, 0, sizeof(int) * number_elements))) return;

    unsigned int block_size = MAX_BLOCK_SIZE / 2;
    unsigned int max_elements_per_block = block_size * 2;
    unsigned int grid_size = number_elements / max_elements_per_block;
    
    if (number_elements % max_elements_per_block != 0) grid_size += 1;

    unsigned int shared_size = max_elements_per_block + ((max_elements_per_block - 1) >> LOG_NUM_BANKS);

    int* block_sums;
    if (checkCudaError(hipMalloc(&block_sums, sizeof(int) * grid_size))) return;
    if (checkCudaError(hipMemset(block_sums, 0, sizeof(int) * grid_size))) return;

    prescan<<<grid_size, block_size, sizeof(int) * shared_size>>>(out, in, max_elements_per_block);

    if (grid_size <= max_elements_per_block) {
        int* dummy_blocks_sums;
        if (checkCudaError(hipMalloc(&dummy_blocks_sums, sizeof(int)))) return;
        if (checkCudaError(hipMemset(dummy_blocks_sums, 0, sizeof(int)))) return;

        prescan<<<1, block_size, sizeof(int) * shared_size>>>(block_sums, block_sums, max_elements_per_block);
        if (checkCudaError(hipFree(dummy_blocks_sums))) return;
    } else {
        int* input_block_sums;
        if (checkCudaError(hipMalloc(&input_block_sums, sizeof(int) * grid_size))) return;
        if (checkCudaError(hipMemcpy(input_block_sums, block_sums, sizeof(int) * grid_size, hipMemcpyDeviceToDevice))) return;

        sum_scan_blelloch(block_sums, input_block_sums, grid_size);
        if (checkCudaError(hipFree(input_block_sums))) return;
    }

    add_block_sums<<<grid_size, block_size>>>(out, out, block_sums, number_elements);
}

__global__ void radix_sort_local(int* out, int* prefix_sums, int* block_sums, unsigned int bit, int* in, unsigned int in_len, unsigned int max_elements_per_block) {
    extern __shared__ int shared_memory[];
    int* shared_data = shared_memory;
    unsigned int mask_out_len = max_elements_per_block + 1;
    int* mask_out = &shared_data[max_elements_per_block];
    int* merged_scan_mask_out = &mask_out[mask_out_len];
    int* mask_out_sums = &merged_scan_mask_out[max_elements_per_block];
    int* scan_mask_out_sums = &mask_out_sums[4];

    unsigned int thread_id = threadIdx.x;

    unsigned int work_id = max_elements_per_block * blockIdx.x + thread_id;
    shared_data[thread_id] = (work_id < in_len) ? in[work_id] : 0;
    
    __syncthreads();

    int thread_data = shared_data[thread_id];
    int extract_bits = (thread_data >> bit) & 3;

    for (unsigned int b = 0; b < 4; b++) {
        mask_out[thread_id] = 0;
        if (thread_id == 0)
            mask_out[mask_out_len - 1] = 0;
        
        __syncthreads();

        bool bits_equals_b = false;
        if (work_id < in_len) {
            bits_equals_b = b == extract_bits;
            mask_out[thread_id] = bits_equals_b;
        }
        __syncthreads();

        int partner = 0;
        int sum = 0;
        for (unsigned int d = 0; d < (unsigned int) log2f(max_elements_per_block); d++) {
            partner = thread_id - (1 << d);
            sum = (partner >= 0) ? mask_out[thread_id] + mask_out[partner] : mask_out[thread_id];
            __syncthreads();
            mask_out[thread_id] = sum;
            __syncthreads();
        }

        int work_val = 0;
        work_val = mask_out[thread_id];
        __syncthreads();
        mask_out[thread_id + 1] = work_val;
        __syncthreads();

        if (thread_id == 0) {
            mask_out[0] = 0;
            int total_sum = mask_out[mask_out_len - 1];
            mask_out_sums[b] = total_sum;
            block_sums[b * gridDim.x + blockIdx.x] = total_sum;
        }
        __syncthreads();

        if (bits_equals_b && (work_id < in_len))
            merged_scan_mask_out[thread_id] = mask_out[thread_id];
        __syncthreads();
    }

    if (thread_id == 0) {
        int run_sum = 0;
        for (unsigned int i = 0; i < 4; i++) {
            scan_mask_out_sums[i] = run_sum;
            run_sum += mask_out_sums[i];
        }
    }

    __syncthreads();

    if (work_id < in_len) {
        int prefix_sum = merged_scan_mask_out[thread_id];
        int new_pos = prefix_sum + scan_mask_out_sums[extract_bits];
        __syncthreads();

        shared_data[new_pos] = thread_data;
        merged_scan_mask_out[new_pos] = prefix_sum;
        __syncthreads();

        prefix_sums[work_id] = merged_scan_mask_out[thread_id];
        out[work_id] = shared_data[thread_id];
    }
}

__global__ void global_shuffle(int* out, int* in, int* scan_block_sums, int* prefix_sums, unsigned int bit, unsigned int in_len, unsigned int max_elements_per_block) {
    unsigned int thread_id = threadIdx.x;
    unsigned int work_id = max_elements_per_block * blockIdx.x + thread_id;

    if (work_id < in_len) {
        int data = in[work_id];
        int extract_bits = (data >> bit) & 3;
        int prefix_sum = prefix_sums[work_id];
        int global_position = scan_block_sums[extract_bits * gridDim.x + blockIdx.x] + prefix_sum;
        __syncthreads();
        out[global_position] = data;
    }
}

void radix_sort(int* const out, int* const in, unsigned int in_len) {
    unsigned int block_size = MAX_BLOCK_SIZE;
    unsigned int max_elements_per_block = block_size;
    unsigned int grid_size = in_len / max_elements_per_block;

    if (in_len % max_elements_per_block != 0)
        grid_size += 1;
    
    int* prefix_sums;
    unsigned int prefix_sums_len = in_len;

    if (checkCudaError(hipMalloc(&prefix_sums, sizeof(int) * prefix_sums_len))) return;
    if (checkCudaError(hipMemset(prefix_sums, 0, sizeof(int) * prefix_sums_len))) return;

    int* block_sums;
    unsigned int block_sums_len = 4 * grid_size;

    if (checkCudaError(hipMalloc(&block_sums, sizeof(int) * block_sums_len))) return;
    if (checkCudaError(hipMemset(block_sums, 0, sizeof(int) * block_sums_len))) return;

    int* scan_block_sums;
    if (checkCudaError(hipMalloc(&scan_block_sums, sizeof(int) * block_sums_len))) return;
    if (checkCudaError(hipMemset(scan_block_sums, 0, sizeof(int) * block_sums_len))) return;

    // unsigned int data_len = max_elements_per_block;
    unsigned int mask_out_len = max_elements_per_block + 1;
    unsigned int merged_scan_mask_out_len = max_elements_per_block;
    unsigned int mask_out_sums_len = max_elements_per_block;
    unsigned int scan_mask_out_sums_len = 4;
    unsigned int shared_size = (mask_out_len + merged_scan_mask_out_len + mask_out_sums_len + scan_mask_out_sums_len) * sizeof(int);

    for (unsigned int bit = 0; bit <= 30; bit += 2) {
        radix_sort_local<<<grid_size, block_size, shared_size>>>(out, prefix_sums, block_sums, bit, in, in_len, max_elements_per_block);
        sum_scan_blelloch(scan_block_sums, block_sums, block_sums_len);
        global_shuffle<<<grid_size, block_size>>>(in, out, scan_block_sums, prefix_sums, bit, in_len, max_elements_per_block);
    }

    if (checkCudaError(hipMemcpy(out, in, sizeof(unsigned int) * in_len, hipMemcpyDeviceToDevice))) return;
    if (checkCudaError(hipFree(scan_block_sums))) return;
    if (checkCudaError(hipFree(block_sums))) return;
    if (checkCudaError(hipFree(prefix_sums))) return;
}

int checkIncremental(int *array, int array_size) {
    for (int i = 0; i < array_size - 1; i++)
        if (array[i] + 1 != array[i + 1]) 
            return false;
    
    return true;    
}

int checkSorted(int *array, int array_size) {
    for (int i = 0; i < array_size - 1; i++)
        if (array[i] > array[i + 1])
            return 0;
    return 1;
}

int main(int argc, char **argv) {
    int array_size = atoi(argv[1]);
    int verbose = atoi(argv[2]);

	srand(time(NULL));
    printf("%d\n", array_size);

    int *to_sort_array;
    int *sorted_array;
    int *input_cuda;
    int *output_cuda;

    // Generate the array to sort
    to_sort_array = (int *) malloc(sizeof(int) * array_size);
    sorted_array = (int *) malloc(sizeof(int) * array_size);
    for (int i = 0; i < array_size; i++)
        to_sort_array[i] = (rand() % 300) + 1; //(array_size - 1) - i; 
    
    // Print the starting array
    if (verbose) {
        printf("Starting array: ");
        for (int i = 0; i < array_size; i++)
            printf("%d ", to_sort_array[i]);
        printf("\n");
    }

    if (checkCudaError(hipMalloc(&input_cuda, sizeof(unsigned int) * array_size))) return -1;
    if (checkCudaError(hipMemcpy(input_cuda, to_sort_array, sizeof(unsigned int) * array_size, hipMemcpyHostToDevice))) return -1;
    if (checkCudaError(hipMalloc(&output_cuda, sizeof(unsigned int) * array_size))) return -1;

    hipEvent_t beginEvent;
    hipEvent_t endEvent;

    hipEventCreate(&beginEvent);
    hipEventCreate(&endEvent);

    hipEventRecord(beginEvent);

    radix_sort(output_cuda, input_cuda, array_size);

    hipEventRecord(endEvent);
    hipEventSynchronize(endEvent);

    float timeValue = 0;
    hipEventElapsedTime(&timeValue, beginEvent, endEvent);
    
    hipEventDestroy(beginEvent);
    hipEventDestroy(endEvent);
    
    if (checkCudaError(hipMemcpy(sorted_array, output_cuda, sizeof(int) * array_size, hipMemcpyDeviceToHost))) return -1;

    // Print the sorted array
    if (verbose) {
        printf("\nRadix sort results: ");
        for (int i = 0; i < array_size; i++)
            printf("%d ", sorted_array[i]);
        printf("\n\n");
    }

    printf("Sorted: %d\n", checkSorted(sorted_array, array_size));
    printf("Incremental: %d\n", checkIncremental(sorted_array, array_size));
    printf("GPU Time: %.2f\n", timeValue);
    delete[] to_sort_array;
    delete[] sorted_array;
    hipFree(input_cuda);
    hipFree(output_cuda);
    hipDeviceReset();
    
    return EXIT_SUCCESS;
}